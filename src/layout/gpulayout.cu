#include "hip/hip_runtime.h"
/*
 * Mycelia immersive 3d network visualization tool.
 * Copyright (C) 2008-2009 Sean Whalen.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <cutil.h>
#include <cutil_math.h>
#include <math_functions.h>
#include <stdio.h>

#define MAX_ITERATIONS 300
#define MAX_DELTA 100
#define COOLING_EXPONENT 1.5
#define VOLUME 1000
#define REPULSION_RADIUS 10000

inline __host__ __device__ void operator+=(float3& p, float4 q)
{
    p.x += q.x;
    p.y += q.y;
    p.z += q.z;
}

inline __host__ __device__ void operator+=(float4& p, float3 q)
{
    p.x += q.x;
    p.y += q.y;
    p.z += q.z;
}

__global__ void
updatePositions(int size, float4* positions_d, float4* positions_delta_from, float4* positions_delta_to, float t)
{
    // sum changes
    int row = threadIdx.x;
    float3 delta;
    
    for(int col = 0; col < size; col++)
    {
        // forces on i computed from i, j
        delta += positions_delta_from[row*size + col];
        
        // forces on i computed from j, i
        delta += positions_delta_to[col*size + row];
    }
    
    // scale if change is too large
    float mag = length(delta);
    
    if(mag > t)
    {
        mag = t / mag;
        delta *= mag;
    }
    
    // update position
    positions_d[row] += delta;
}

__global__ void
calculateForces(int size, float4* positions_d, float4* positions_delta_from, float4* positions_delta_to, int* adjacencies_d, float t, float k)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i == j || i >= size || j >= size) return;
    
    // repel
    float3 v = make_float3(positions_d[i]) - make_float3(positions_d[j]);
    float mag = length(v);
    v /= mag;
    float f = k*k * (1/mag - mag*mag/REPULSION_RADIUS) * positions_d[i].w;
    
    // update relative change
    int offset = i*size + j;
    positions_delta_from[offset] += v * f;
    positions_delta_to[offset] += v * -f;
    
    // attract if connected
    f = (mag*mag/k) * adjacencies_d[offset]; // avoid conditional by setting force to 0 if not adjacent
    
    // update relative change
    positions_delta_from[offset] += v * -f;
    positions_delta_to[offset] += v * f;
}

extern "C"
{
    __host__ void
    gpuLayout(float4* positions_h, int* adjacencies_h, int size)
    {
        /*int device;
        struct hipDeviceProp_t prop;
        CUDA_SAFE_CALL(hipGetDevice(&device));
        CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, device));
        printf("%d %s %d.%d\n", device, prop.name, prop.major, prop.minor);*/
        
        float k = pow(VOLUME/(float)size, 1/3.0f);
        dim3 dimBlock(20, 20);
        dim3 dimGrid((size+dimBlock.x-1) / dimBlock.x, (size+dimBlock.y-1) / dimBlock.y);
        
        float4* positions_d;
        CUDA_SAFE_CALL(hipMalloc((void**)&positions_d, sizeof(float4)*size));
        hipMemcpy(positions_d, positions_h, sizeof(float4)*size, hipMemcpyHostToDevice);
        
        float4* positions_delta_from;
        CUDA_SAFE_CALL(hipMalloc((void**)&positions_delta_from, sizeof(float4)*size*size));
        
        float4* positions_delta_to;
        CUDA_SAFE_CALL(hipMalloc((void**)&positions_delta_to, sizeof(float4)*size*size));
        
        int* adjacencies_d;
        CUDA_SAFE_CALL(hipMalloc((void**)&adjacencies_d, sizeof(int)*size*size));
        hipMemcpy(adjacencies_d, adjacencies_h, sizeof(int)*size*size, hipMemcpyHostToDevice);
        
        for(int i = MAX_ITERATIONS; i >= 0; i--)
        {
            float t = MAX_DELTA * pow(i/(double)MAX_ITERATIONS, COOLING_EXPONENT);
            
            hipMemset(positions_delta_from, 0, sizeof(float4)*size*size);
            hipMemset(positions_delta_to, 0, sizeof(float4)*size*size);
            
            calculateForces<<<dimGrid, dimBlock>>>(size, positions_d, positions_delta_from, positions_delta_to, adjacencies_d, t, k);
            updatePositions<<<1, size>>>(size, positions_d, positions_delta_from, positions_delta_to, t);
            
            hipDeviceSynchronize();
        }
        
        hipMemcpy(positions_h, positions_d, sizeof(float4)*size, hipMemcpyDeviceToHost);
        hipFree(positions_d);
        hipFree(positions_delta_from);
        hipFree(positions_delta_to);
        hipFree(adjacencies_d);
    }
}